#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define STB_IMAGE_IMPLEMENTATION
#include "/home/bowen/CUDAEindoefening/stb/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "/home/bowen/CUDAEindoefening/stb/stb_image_write.h"

// Convolution
__global__ void convolution_cuda(unsigned char* input, unsigned char* output, int width, int height, int channels, const int* kernel, int kernel_size) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int half_k = kernel_size / 2;

    if (x >= width || y >= height) return;

    for (int c = 0; c < channels; c++) {
        int sum = 0;
        for (int ky = -half_k; ky <= half_k; ky++) {
            for (int kx = -half_k; kx <= half_k; kx++) {
                int ix = min(max(x + kx, 0), width - 1);
                int iy = min(max(y + ky, 0), height - 1);
                int pixel = input[(iy * width + ix) * channels + c];
                int kval = kernel[(ky + half_k) * kernel_size + (kx + half_k)];
                sum += pixel * kval;
            }
        }
        sum = min(max(sum, 0), 255);
        output[(y * width + x) * channels + c] = (unsigned char)sum;
    }
}

// Max Pooling
__global__ void max_pooling(unsigned char* input, unsigned char* output, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int out_width = width / 2;
    int out_height = height / 2;

    if (x >= out_width || y >= out_height) return;

    for (int c = 0; c < channels; c++) {
        unsigned char max_val = 0;
        for (int ky = 0; ky < 2; ky++) {
            for (int kx = 0; kx < 2; kx++) {
                int ix = x * 2 + kx;
                int iy = y * 2 + ky;
                unsigned char pixel = input[(iy * width + ix) * channels + c];
                max_val = max(max_val, pixel);
            }
        }
        output[(y * out_width + x) * channels + c] = max_val;
    }
}

// Min Pooling
__global__ void min_pooling(unsigned char* input, unsigned char* output, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int out_width = width / 2;
    int out_height = height / 2;

    if (x >= out_width || y >= out_height) return;

    for (int c = 0; c < channels; c++) {
        unsigned char min_val = 255;
        for (int ky = 0; ky < 2; ky++) {
            for (int kx = 0; kx < 2; kx++) {
                int ix = x * 2 + kx;
                int iy = y * 2 + ky;
                unsigned char pixel = input[(iy * width + ix) * channels + c];
                min_val = min(min_val, pixel);
            }
        }
        output[(y * out_width + x) * channels + c] = min_val;
    }
}

// Average Pooling
__global__ void average_pooling(unsigned char* input, unsigned char* output, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int out_width = width / 2;
    int out_height = height / 2;

    if (x >= out_width || y >= out_height) return;

    for (int c = 0; c < channels; c++) {
        int sum = 0;
        for (int ky = 0; ky < 2; ky++) {
            for (int kx = 0; kx < 2; kx++) {
                int ix = x * 2 + kx;
                int iy = y * 2 + ky;
                sum += input[(iy * width + ix) * channels + c];
            }
        }
        output[(y * out_width + x) * channels + c] = sum / 4;
    }
}

int main() {
    const char* input_path = "input.png";
    int width, height, channels;

    // === Load input image ===
    unsigned char* input_image = stbi_load(input_path, &width, &height, &channels, 0);
    if (!input_image) {
        printf("Failed to load image.\n");
        return 1;
    }

    // === Strip alpha channel if present ===
    if (channels == 4) {
        unsigned char* rgb_image = (unsigned char*)malloc(width * height * 3);
        for (int i = 0; i < width * height; i++) {
            rgb_image[i * 3 + 0] = input_image[i * 4 + 0];
            rgb_image[i * 3 + 1] = input_image[i * 4 + 1];
            rgb_image[i * 3 + 2] = input_image[i * 4 + 2];
        }
        free(input_image);
        input_image = rgb_image;
        channels = 3;
    }

    // === Kernel Definition ===
    const int kernelSize = 3;
    int host_kernel[kernelSize * kernelSize] = {
        1, 0, -1,
        1, 0, -1,
        1, 0, -1
    };

    // === Allocate memory sizes ===
    size_t img_size = width * height * channels * sizeof(unsigned char);
    size_t pooled_width = width / 2;
    size_t pooled_height = height / 2;
    size_t pooled_img_size = pooled_width * pooled_height * channels * sizeof(unsigned char);

    // === Device memory ===
    unsigned char *d_input;
    unsigned char *d_output_conv, *d_output_max, *d_output_min, *d_output_avg;
    int* d_kernel;

    hipMalloc(&d_input, img_size);
    hipMalloc(&d_output_conv, img_size);
    hipMalloc(&d_output_max, pooled_img_size);
    hipMalloc(&d_output_min, pooled_img_size);
    hipMalloc(&d_output_avg, pooled_img_size);
    hipMalloc(&d_kernel, kernelSize * kernelSize * sizeof(int));

    hipMemcpy(d_input, input_image, img_size, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, host_kernel, kernelSize * kernelSize * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockDim(32, 32);
    dim3 gridDim_full((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);
    dim3 gridDim_pooled((pooled_width + blockDim.x - 1) / blockDim.x, (pooled_height + blockDim.y - 1) / blockDim.y);

    // === Host result buffers ===
    unsigned char* result_conv = (unsigned char*)malloc(img_size);
    unsigned char* result_max  = (unsigned char*)malloc(pooled_img_size);
    unsigned char* result_min  = (unsigned char*)malloc(pooled_img_size);
    unsigned char* result_avg  = (unsigned char*)malloc(pooled_img_size);

    // === Create CUDA streams ===
    hipStream_t stream1, stream2, stream3, stream4;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
    hipStreamCreate(&stream4);
    
    // === CONVOLUTION ===
    convolution_cuda<<<gridDim_full, blockDim, 0, stream1>>>(d_input, d_output_conv, width, height, channels, d_kernel, kernelSize);

    // === MAX POOLING ===
    max_pooling<<<gridDim_pooled, blockDim, 0, stream2>>>(d_input, d_output_max, width, height, channels);

    // === MIN POOLING ===
    min_pooling<<<gridDim_pooled, blockDim, 0, stream3>>>(d_input, d_output_min, width, height, channels);

    // === AVERAGE POOLING ===
    average_pooling<<<gridDim_pooled, blockDim, 0, stream4>>>(d_input, d_output_avg, width, height, channels);

    // === Synchronize streams ===
    hipDeviceSynchronize();

    // === Copy results to host ===
    hipMemcpy(result_conv, d_output_conv, img_size, hipMemcpyDeviceToHost);
    hipMemcpy(result_max,  d_output_max,  pooled_img_size, hipMemcpyDeviceToHost);
    hipMemcpy(result_min,  d_output_min,  pooled_img_size, hipMemcpyDeviceToHost);
    hipMemcpy(result_avg,  d_output_avg,  pooled_img_size, hipMemcpyDeviceToHost);

    // === Save results ===
    stbi_write_png("convolution.png", width, height, channels, result_conv, width * channels);
    stbi_write_png("max_pooling.png", pooled_width, pooled_height, channels, result_max, pooled_width * channels);
    stbi_write_png("min_pooling.png", pooled_width, pooled_height, channels, result_min, pooled_width * channels);
    stbi_write_png("average_pooling.png", pooled_width, pooled_height, channels, result_avg, pooled_width * channels);

    // === Output ===
    printf("Afbeeldingen opgeslagen:\n");
    printf(" - convolution.png\n");
    printf(" - max_pooling.png\n");
    printf(" - min_pooling.png\n");
    printf(" - average_pooling.png\n");

    // === Cleanup ===
    stbi_image_free(input_image);
    free(result_conv);
    free(result_max);
    free(result_min);
    free(result_avg);

    hipFree(d_input);
    hipFree(d_output_conv);
    hipFree(d_output_max);
    hipFree(d_output_min);
    hipFree(d_output_avg);
    hipFree(d_kernel);

    // === Destroy streams ===
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);
    hipStreamDestroy(stream4);

    return 0;
}
