#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define STB_IMAGE_IMPLEMENTATION
#include "/home/bowen/CUDAEindoefening/stb/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "/home/bowen/CUDAEindoefening/stb/stb_image_write.h"

// Convolution
__global__ void convolution_cuda(unsigned char* input, unsigned char* output, int width, int height, int channels, const int* kernel, int kernel_size) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int half_k = kernel_size / 2;

    if (x >= width || y >= height) return;

    for (int c = 0; c < channels; c++) {
        int sum = 0;
        for (int ky = -half_k; ky <= half_k; ky++) {
            for (int kx = -half_k; kx <= half_k; kx++) {
                int ix = min(max(x + kx, 0), width - 1);
                int iy = min(max(y + ky, 0), height - 1);
                int pixel = input[(iy * width + ix) * channels + c];
                int kval = kernel[(ky + half_k) * kernel_size + (kx + half_k)];
                sum += pixel * kval;
            }
        }
        sum = min(max(sum, 0), 255);
        output[(y * width + x) * channels + c] = (unsigned char)sum;
    }
}

// Max Pooling
__global__ void max_pooling(unsigned char* input, unsigned char* output, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int out_width = width / 2;
    int out_height = height / 2;

    if (x >= out_width || y >= out_height) return;

    for (int c = 0; c < channels; c++) {
        unsigned char max_val = 0;
        for (int ky = 0; ky < 2; ky++) {
            for (int kx = 0; kx < 2; kx++) {
                int ix = x * 2 + kx;
                int iy = y * 2 + ky;
                unsigned char pixel = input[(iy * width + ix) * channels + c];
                max_val = max(max_val, pixel);
            }
        }
        output[(y * out_width + x) * channels + c] = max_val;
    }
}

// Min Pooling
__global__ void min_pooling(unsigned char* input, unsigned char* output, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int out_width = width / 2;
    int out_height = height / 2;

    if (x >= out_width || y >= out_height) return;

    for (int c = 0; c < channels; c++) {
        unsigned char min_val = 255;
        for (int ky = 0; ky < 2; ky++) {
            for (int kx = 0; kx < 2; kx++) {
                int ix = x * 2 + kx;
                int iy = y * 2 + ky;
                unsigned char pixel = input[(iy * width + ix) * channels + c];
                min_val = min(min_val, pixel);
            }
        }
        output[(y * out_width + x) * channels + c] = min_val;
    }
}

// Average Pooling
__global__ void average_pooling(unsigned char* input, unsigned char* output, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int out_width = width / 2;
    int out_height = height / 2;

    if (x >= out_width || y >= out_height) return;

    for (int c = 0; c < channels; c++) {
        int sum = 0;
        for (int ky = 0; ky < 2; ky++) {
            for (int kx = 0; kx < 2; kx++) {
                int ix = x * 2 + kx;
                int iy = y * 2 + ky;
                sum += input[(iy * width + ix) * channels + c];
            }
        }
        output[(y * out_width + x) * channels + c] = sum / 4;
    }
}

int main() {
    const char* input_path = "input.png";
    int width, height, channels;

    // === Load input image ===
    unsigned char* input_image = stbi_load(input_path, &width, &height, &channels, 0);
    if (!input_image) {
        printf("Failed to load image.\n");
        return 1;
    }

    // === Initialize CUDA context to reduce timing spikes ===
    hipFree(0);=

    // === Strip alpha channel if present ===
    if (channels == 4) {
        unsigned char* rgb_image = (unsigned char*)malloc(width * height * 3);
        for (int i = 0; i < width * height; i++) {
            rgb_image[i * 3 + 0] = input_image[i * 4 + 0];
            rgb_image[i * 3 + 1] = input_image[i * 4 + 1];
            rgb_image[i * 3 + 2] = input_image[i * 4 + 2];
        }
        free(input_image);
        input_image = rgb_image;
        channels = 3;
    }

    // === Kernel Definition ===
    const int kernelSize = 3;
    int host_kernel[kernelSize * kernelSize] = {
        1, 0, -1,
        1, 0, -1,
        1, 0, -1
    };

    // === Allocate memory sizes ===
    size_t img_size = width * height * channels * sizeof(unsigned char);
    size_t pooled_width = width / 2;
    size_t pooled_height = height / 2;
    size_t pooled_img_size = pooled_width * pooled_height * channels * sizeof(unsigned char);

    // === Device memory ===
    unsigned char *d_input, *d_output_full, *d_output_pooled;
    int* d_kernel;

    hipMalloc(&d_input, img_size);
    hipMalloc(&d_output_full, img_size);
    hipMalloc(&d_output_pooled, pooled_img_size);
    hipMalloc(&d_kernel, kernelSize * kernelSize * sizeof(int));

    hipMemcpy(d_input, input_image, img_size, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, host_kernel, kernelSize * kernelSize * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockDim(32, 32);
    dim3 gridDim_full((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);
    dim3 gridDim_pooled((pooled_width + blockDim.x - 1) / blockDim.x, (pooled_height + blockDim.y - 1) / blockDim.y);

    // === Host result buffers ===
    unsigned char* result_full = (unsigned char*)malloc(img_size);
    unsigned char* result_pooled = (unsigned char*)malloc(pooled_img_size);

    // === CONVOLUTION ===
    convolution_cuda<<<gridDim_full, blockDim>>>(d_input, d_output_full, width, height, channels, d_kernel, kernelSize);
    hipDeviceSynchronize();
    hipMemcpy(result_full, d_output_full, img_size, hipMemcpyDeviceToHost);
    stbi_write_png("convolution.png", width, height, channels, result_full, width * channels);

    // === MAX POOLING ===
    max_pooling<<<gridDim_pooled, blockDim>>>(d_input, d_output_pooled, width, height, channels);
    hipDeviceSynchronize();
    hipMemcpy(result_pooled, d_output_pooled, pooled_img_size, hipMemcpyDeviceToHost);
    stbi_write_png("max_pooling.png", pooled_width, pooled_height, channels, result_pooled, pooled_width * channels);

    // === MIN POOLING ===
    min_pooling<<<gridDim_pooled, blockDim>>>(d_input, d_output_pooled, width, height, channels);
    hipDeviceSynchronize();
    hipMemcpy(result_pooled, d_output_pooled, pooled_img_size, hipMemcpyDeviceToHost);
    stbi_write_png("min_pooling.png", pooled_width, pooled_height, channels, result_pooled, pooled_width * channels);

    // === AVERAGE POOLING ===
    average_pooling<<<gridDim_pooled, blockDim>>>(d_input, d_output_pooled, width, height, channels);
    hipDeviceSynchronize();
    hipMemcpy(result_pooled, d_output_pooled, pooled_img_size, hipMemcpyDeviceToHost);
    stbi_write_png("average_pooling.png", pooled_width, pooled_height, channels, result_pooled, pooled_width * channels);

    // === Output ===
    printf("Afbeeldingen opgeslagen:\n");
    printf(" - convolution.png\n");
    printf(" - max_pooling.png\n");
    printf(" - min_pooling.png\n");
    printf(" - average_pooling.png\n");

    // === Cleanup ===
    stbi_image_free(input_image);
    free(result_full);
    free(result_pooled);
    hipFree(d_input);
    hipFree(d_output_full);
    hipFree(d_output_pooled);
    hipFree(d_kernel);

    return 0;
}
