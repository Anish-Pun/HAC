#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define STB_IMAGE_IMPLEMENTATION
#include "/home/bowen/CUDAEindoefening/stb/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "/home/bowen/CUDAEindoefening/stb/stb_image_write.h"

// convolution
__global__ void convolution_cuda(unsigned char* input, unsigned char* output, int width, int height, int channels, const int* kernel, int kernel_size) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int half_k = kernel_size / 2;

    if (x >= width || y >= height) return;

    for (int c = 0; c < channels; c++) {
        int sum = 0;
        for (int ky = -half_k; ky <= half_k; ky++) {
            for (int kx = -half_k; kx <= half_k; kx++) {
                int ix = min(max(x + kx, 0), width - 1);
                int iy = min(max(y + ky, 0), height - 1);
                int pixel = input[(iy * width + ix) * channels + c];
                int kval = kernel[(ky + half_k) * kernel_size + (kx + half_k)];
                sum += pixel * kval;
            }
        }
        sum = min(max(sum, 0), 255);
        output[(y * width + x) * channels + c] = (unsigned char)sum;
    }
}

// Max Pooling
__global__ void max_pooling(unsigned char* input, unsigned char* output, int width, int height, int channels, int pool_size) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int half_pool = pool_size / 2;

    if (x >= width || y >= height) return;

    for (int c = 0; c < channels; c++) {
        unsigned char max_val = 0;
        for (int ky = -half_pool; ky <= half_pool; ky++) {
            for (int kx = -half_pool; kx <= half_pool; kx++) {
                int ix = min(max(x + kx, 0), width - 1);
                int iy = min(max(y + ky, 0), height - 1);
                unsigned char pixel = input[(iy * width + ix) * channels + c];
                max_val = max(max_val, pixel);
            }
        }
        output[(y * width + x) * channels + c] = max_val;
    }
}

// Min Pooling
__global__ void min_pooling(unsigned char* input, unsigned char* output, int width, int height, int channels, int pool_size) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int half_pool = pool_size / 2;

    if (x >= width || y >= height) return;

    for (int c = 0; c < channels; c++) {
        unsigned char min_val = 255;
        for (int ky = -half_pool; ky <= half_pool; ky++) {
            for (int kx = -half_pool; kx <= half_pool; kx++) {
                int ix = min(max(x + kx, 0), width - 1);
                int iy = min(max(y + ky, 0), height - 1);
                unsigned char pixel = input[(iy * width + ix) * channels + c];
                min_val = min(min_val, pixel);
            }
        }
        output[(y * width + x) * channels + c] = min_val;
    }
}

// Average Pooling
__global__ void average_pooling(unsigned char* input, unsigned char* output, int width, int height, int channels, int pool_size) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int half_pool = pool_size / 2;

    if (x >= width || y >= height) return;

    for (int c = 0; c < channels; c++) {
        int sum = 0;
        int count = 0;
        for (int ky = -half_pool; ky <= half_pool; ky++) {
            for (int kx = -half_pool; kx <= half_pool; kx++) {
                int ix = min(max(x + kx, 0), width - 1);
                int iy = min(max(y + ky, 0), height - 1);
                sum += input[(iy * width + ix) * channels + c];
                count++;
            }
        }
        unsigned char avg_val = sum / count;
        output[(y * width + x) * channels + c] = avg_val;
    }
}

int main() {
    const char* input_path = "input.png";
    int width, height, channels;
    unsigned char* input_image = stbi_load(input_path, &width, &height, &channels, 0);
    if (!input_image) {
        printf("Failed to load image.\n");
        return 1;
    }

    const int kernelWidth = 3;
    const int kernelHeight = 3;
    int host_kernel[kernelHeight * kernelWidth] = {
        1, 0, -1,
        1, 0, -1,
        1, 0, -1
    };

    size_t img_size = width * height * channels * sizeof(unsigned char);
    unsigned char *d_input, *d_output, *d_pool_output;
    int* d_kernel;

    hipMalloc((void**)&d_input, img_size);
    hipMalloc((void**)&d_output, img_size);
    hipMalloc((void**)&d_pool_output, img_size);
    hipMalloc((void**)&d_kernel, kernelWidth * kernelHeight * sizeof(int));

    hipMemcpy(d_input, input_image, img_size, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, host_kernel, kernelWidth * kernelHeight * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((width + 15) / 16, (height + 15) / 16);

    // Perform Convolution
    convolution_cuda<<<gridDim, blockDim>>>(d_input, d_output, width, height, channels, d_kernel, kernelWidth);
    hipDeviceSynchronize();

    // === CONVOLUTION ===
    unsigned char* convolution_result = (unsigned char*)malloc(img_size);
    hipMemcpy(convolution_result, d_output, img_size, hipMemcpyDeviceToHost);
    stbi_write_jpg("convolution.jpg", width, height, channels, convolution_result, 90);

    // Allocate host memory for pooling results
    unsigned char* result_image = (unsigned char*)malloc(img_size);

    // === MAX POOLING ===
    max_pooling<<<gridDim, blockDim>>>(d_input, d_pool_output, width, height, channels, 2);
    hipMemcpy(result_image, d_pool_output, img_size, hipMemcpyDeviceToHost);
    stbi_write_jpg("max_pooling.jpg", width, height, channels, result_image, 90);

    // === MIN POOLING ===
    min_pooling<<<gridDim, blockDim>>>(d_input, d_pool_output, width, height, channels, 2);
    hipMemcpy(result_image, d_pool_output, img_size, hipMemcpyDeviceToHost);
    stbi_write_jpg("min_pooling.jpg", width, height, channels, result_image, 90);

    // === AVERAGE POOLING ===
    average_pooling<<<gridDim, blockDim>>>(d_input, d_pool_output, width, height, channels, 2);
    hipMemcpy(result_image, d_pool_output, img_size, hipMemcpyDeviceToHost);
    stbi_write_jpg("average_pooling.jpg", width, height, channels, result_image, 90);

    printf("Afbeeldingen opgeslagen:\n");
    printf(" - convolution.jpg\n");
    printf(" - max_pooling.jpg\n");
    printf(" - min_pooling.jpg\n");
    printf(" - average_pooling.jpg\n");

    // Cleanup
    stbi_image_free(input_image);
    free(result_image);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_pool_output);
    hipFree(d_kernel);

    return 0;
}
